#include "hip/hip_runtime.h"
#include "Mandelbrot.h"
#include <cmath>
#include <cstdio>
#include <iostream>
#include "hip/hip_runtime.h"
#include ""

using namespace std;

#define complexPlaneWidth 2.0
#define complexPlaneLeftEdge -1.5
#define complexPlaneHeight 2.0
#define complexPlaneBottomEdge -1.0

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

namespace{
	__device__ double lerp(double v0, double v1, double t) {
		return v0+(v1-v0)*t;
	}
	__device__ double dist(double count, double x, double y) {
		double zn = sqrt(x + y);
		double mu = logf(logf(zn))/logf(2);
		return count + 1 -mu;
	}
	/*bool checkBulb(double x, double y2) {
		double q = (x - 0.25)*(x - 0.25) - y2;
		return ((4 * q * (q +(x -0.25))<y2) || (16*((x+1)*(x+1)+y2 < 1)));
	}*/
	// Takes in an x pixel coordinate and converts it into the corresponding
	// real coordinate on the complex plane
	__device__ double computeRealFromX(unsigned int x, unsigned int width) {
		// x in the range(0, bitmapwidth)
		double f = (double)x / width;

		// remap x to range(0, complexPlaneWidth)
		double realPart = (f*complexPlaneWidth) + complexPlaneLeftEdge;

		return realPart;
	}

	// Tkaes in a y pixel coordinate and convertes it into the correpsonding
	// imaginary coordinate on the complex plane
	__device__ double computeImaginaryFromY(unsigned int y, unsigned int height) {
		double f = (double)y / height;
		double imaginaryPart = (f*complexPlaneHeight) + complexPlaneBottomEdge;
		return imaginaryPart;
	}

	__device__ void setColor(unsigned char* img, unsigned int x, unsigned int y, unsigned int width, unsigned char red, unsigned char green, unsigned char blue) {
		int offset = (y*width + x)*4;
		img[offset] = red;
		img[offset+1] = green;
		img[offset+2] = blue;
		img[offset+3] = 255;
	}

	//Convert from hsv to rgb values
	__device__ void hsvToRgb(double h, double s, double v, unsigned char rgb[]) {
		double r, g, b;

		int i = (int)(h*6);
		double f = h * 6 - i;
		double p = v * (1 - s);
		double q = v * (1 - f * s);
		double t = v * (1 - (1 - f) * s);

		switch(i % 6){
		case 0: r = v, g = t, b = p; break;
		case 1: r = q, g = v, b = p; break;
		case 2: r = p, g = v, b = t; break;
		case 3: r = p, g = q, b = v; break;
		case 4: r = t, g = p, b = v; break;
		case 5: r = v, g = p, b = q; break;
		}

		rgb[0] = r * 255;
		rgb[1] = g * 255;
		rgb[2] = b * 255;
	}

	__global__ void complex_coord(double* coord, unsigned int width, unsigned int height) {
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		int j = blockIdx.y * blockDim.y + threadIdx.y;

		if( i >= width || j >= height)
			return;

		//Convert from screen coordinates to complex coordinates
		double x0 = computeRealFromX(i, width);
		double y0 = computeImaginaryFromY(j, height);

		int idx = j*width + i;
		coord[idx] = x0;
		coord[idx+1] = y0;
	}
	__global__ void fractal_kernel(double* coord, double* iters, unsigned int width, unsigned int height) {
		//Start z at 0
				int i = blockIdx.x * blockDim.x + threadIdx.x;
		int j = blockIdx.y * blockDim.y + threadIdx.y;

		if( i >= width || j >= height)
			return;
		int idx = j*width+i;

		double x0 = coord[idx];
		double y0 = coord[idx+1];
		double x = 0;
		double y = 0;
		double x2 = 0.0;
		double y2 = 0.0;

		//Iterate until we reach 2 or the max
		int count = 0;
		for(count = 0; count < MAXITER && ((x2 + y2) < 4.0); count++) {
			double xtemp = x2 - y2 + x0;
			y = 2*x*y + y0;
			x = xtemp;
			y2 = y*y;
			x2 = x*x;
		}
		coord[idx] = x2;
		coord[idx+1] = y2;
		iters[idx] = (double)count;
	}
	__global__ void color_fractal(unsigned char* img, double* coord, double* iters, unsigned int width, unsigned int height) {
		int i = blockIdx.x * blockDim.x + threadIdx.x;
		int j = blockIdx.y * blockDim.y + threadIdx.y;
		if( i >= width || j >= height)
			return;
		int idx = j*width+i;

		double iter = iters[idx];
		double x2 = coord[idx];
		double y2 = coord[idx+1];

		unsigned char rgb[3];

		//Color the pixel
		if ((unsigned int)iter == MAXITER) {
			setColor(img, i, j, width, 0, 0, 0);
		}
		else {
			iter = dist(iter, x2, y2);
			//Create the color palatte (0x0, 0xffffff)
			double color = lerp(floor(iter), floor(iter+1), fmod(iter, 1.0));
			double hue = 0.95 * 10 * (color/MAXITER);
			hsvToRgb(hue, 0.6, 1.0, rgb);
			setColor(img, i, j, width, rgb[0], rgb[1], rgb[2]);
		}
	}
}

// Keep track of the complex coordinates and the iteration count in multiple kernels
void Mandelbrot::gen_fractal()
{
	unsigned int dimx = get_width();
	unsigned int dimy = get_height();
	unsigned int size = dimx*dimy;
	unsigned int num_bytes = size*4*sizeof(unsigned char);

	double* d_complex;
	double* d_iters;
	cudaErrorCheck(hipMalloc((void**)&d_complex, size*sizeof(double)));
	cudaErrorCheck(hipMalloc((void**)&d_iters, size*sizeof(double)));
	unsigned char* d_a=0;
	cudaErrorCheck(hipMalloc((void**)&d_a, num_bytes));
	if(0==d_a || 0 == d_complex || 0 == d_iters) {
		cout << "Could not allocate memory" << endl;
		return;
	}

	cudaErrorCheck(hipMemset(d_a, 0, num_bytes));

	dim3 grid, block;
	block.x = 16;
	grid.x = dimx/block.x;

	block.y = 16;
	grid.y = dimy/block.y;

	complex_coord<<<grid, block>>>(d_complex, dimx, dimy);
	fractal_kernel<<<grid, block>>>(d_complex, d_iters, dimx, dimy);
	color_fractal<<<grid, block>>>(d_a, d_complex, d_iters, dimx, dimy);

	cudaErrorCheck(hipMemcpy(m_bitmap, d_a, num_bytes,hipMemcpyDeviceToHost));
	
	hipFree(d_a);
	hipFree(d_iters);
	hipFree(d_complex);
}